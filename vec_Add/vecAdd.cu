
#include <hip/hip_runtime.h>
#include<stdio.h>

// Kernel definition
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
int i = blockDim.x*blockIdx.x + threadIdx.x;

if(i<N) 
	C[i] = A[i] + B[i];

printf("In thread-i, we are using value %f + %f  = %f\n", A[i], B[i], C[i]);
}

int main()
{
int N = 1024;
float* h_A, *h_B, *h_C;

size_t arrSize = N*sizeof(float);

h_A = (float*)malloc(arrSize);
h_B = (float*)malloc(arrSize);
h_C = (float*)malloc(arrSize);


for(int i=0;i<N;i++)
	h_A[i] = h_B[i] = i;


float *d_A,*d_B, *d_C;
hipMalloc(&d_A, arrSize);
hipMalloc(&d_B, arrSize);
hipMalloc(&d_C, arrSize);

// Copy vectors from host memory to device memory
hipMemcpy(d_A, h_A, arrSize, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_A, arrSize, hipMemcpyHostToDevice);

int threadsPerBlock = 256;
int blocksPerGrid = N/threadsPerBlock;
if(N%threadsPerBlock) blocksPerGrid++;

//(N + threadsPerBlock – 1) / threadsPerBlock;


VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

hipMemcpy(h_C, d_C, arrSize, hipMemcpyDeviceToHost);

for(int i=0;i<N;i++)
	printf("%f %f %f\n", h_A[i], h_B[i], h_C[i]);

hipFree(d_A);
hipFree(d_B);
hipFree(d_C);

return 0;
}


